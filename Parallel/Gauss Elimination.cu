
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>
#include<time.h>
using namespace std;
using namespace std::chrono;

int power(int x,int n,int m){
    if (n==0)
        return 1;
    int p=power(x,n/2,m)%m;
    p=(p*p)%m;
    if (n%2==0)
        return p;
    else
        return (x*p)%m;
    }

int modArthInv(int a,int m){
    return power(a,m-2,m);
}

__global__
void rre(int i, int j, int n, int *mat, int p, int m)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
  	//int stride = blockDim.x * gridDim.x;
	//for(; k+i <= n; k+=stride)
	//{
		//mat[j*(n+1) + k+i] -= p * mat[i*(n+1) + k+i];
		//mat[j*(n+1) + k+i] %= m;
		//if (mat[j*(n+1) + k+i] < 0) {
			//mat[j*(n+1) + k+i] += m;
		//}
	//}
	if(k+i <= n)
	{
		mat[j*(n+1) + k+i] -= p * mat[i*(n+1) + k+i];
		mat[j*(n+1) + k+i] %= m;
		if (mat[j*(n+1) + k+i] < 0) {
			mat[j*(n+1) + k+i] += m;
		}
	}
}

int main() {
	hipError_t err = hipSuccess;
	int n = 1000, m=1009, t;
	cout<<n<<m<<endl;
	//cin >> n >> m;
	int *mat;
	err = hipMallocManaged((void **)&mat, n*(n+1)*sizeof(int));
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	//unordered_map
	srand(time(0));
	int i, j, k;
	for (i = 0; i < n; i++)
	{
		//vector<int> temp;
		for (j = 0; j <= n; j++)
		{
			//cin >> t;
			t = rand() % m;
			//cout << t << " ";
			//temp.push_back(t);
			mat[i*(n+1) + j] = t;
		}
		//cout << "\n";
		//mat.push_back(temp);
	}

	auto start = high_resolution_clock::now();

	for (i = 0; i < n; i++) {
		//cout<<i<<endl;
		if (mat[i*(n+1) + i] == 0) {
			for (j = i + 1; j < n; j++) {
				if (mat[j*(n+1) + i] != 0) {
					//swaprows(mat, i, j, n);
					//vector<int> temp = mat[j];
					//mat[j] = mat[i];
					//mat[i] = temp;
					for(k = 0; k <=n; k++)
					{
						int t = mat[i*(n+1) + k];
						mat[i*(n+1) + k] = mat[j*(n+1) + k];
						mat[j*(n+1) + k] = t;
					}
					break;
				}
			}
		}
		if (mat[i*(n+1) + i] == 0) {
			cout << "No Solution";
			exit(0);
		}
		else {
			if (mat[i*(n+1) + i] != 1) {
				int inv = modArthInv(m, mat[i*(n+1) + i]);
				for (j = 0; j <= n; j++) {
					mat[i*(n+1) + j] *= inv;
					mat[i*(n+1) + j] %= m;
					if (mat[i*(n+1) + j] < 0) {
						mat[i*(n+1) + j] += m;
					}
				}
			}

			for (j = 0; j < n; j++) {
				if(j !=i && mat[j*(n+1) + i] != 0)
				{
					rre<<<(n + 1023)/1024,1024>>>(i, j, n, mat, mat[j*(n+1) + i], m);
					hipDeviceSynchronize();
				}
			}

			/*
			for (j = 0; j < n; j++) {
				if(j != i && mat[j][i] != 0) {
					int temp = mat[j][i];
					for (k = i; k <= n; k++) {
						mat[j][k] -= temp * mat[i][k];
						mat[j][k] %= m;
						if (mat[j][k] < 0) {
							mat[j][k] += m;
						}
					}			} */
		}
	}

	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);

	/*
	for (i = 0; i < n; i++) {
		cout << mat[i*(n+1) + n]%m << "\n";
	}
	*/

	cout << "Time: "<<duration.count()<<endl;

	hipFree(mat);
	return 0;
}
